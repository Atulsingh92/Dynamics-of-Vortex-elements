#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

__global__
void NewVortexDistrub(float *V, float *O, float *VN, float *ON, float *S, int N) { // the kernel carries out the assigned formulas on each 
											// thread. i.e. each thread carries the following program.
	__shared__ float Vc[3];			// This kernel calculates angular velocity , Energy, Maximum speed and radius of vortex 
	__shared__ float dVx[3];
	__shared__ float dVy[3];		// Shared variables to give threads access to these variables.
	__shared__ float dVz[3];
	__shared__ float domdt[3];

 int tx = threadIdx.x;			// cuda inbuilit variable threadIdx.x keeps the track of thread id in a block, is assigned to tx
 int bx =  blockIdx.x;			// cuda inbuilt variable blockIdx.x keeps track of blocks assigned during the kernel call
float radiika, dssss_dr;
float ssss;
float t1,t2,t3;

	if (tx<3) {					// Initializes the variable
		if (tx == 0) {
			Vc[tx] = 1.0f;
		}
		else {
			Vc[tx] = 0.0f;
			dVx[tx] = 0.0f;
			dVy[tx] = 0.0f;
			dVz[tx] = 0.0f;
		}
	}

    t1 = ((V[(bx * 3) + 1] - V[(tx * 3) + 1]) * O[(tx * 3) + 2] - (V[(bx * 3) + 2] - V[(tx * 3) + 2]) * O[(tx * 3) + 1]);
    t2 = ((V[(bx * 3) + 2] - V[(tx * 3) + 2]) * O[(tx * 3 + 0)] - (V[(bx * 3) + 0] - V[(tx * 3) + 0]) * O[(tx * 3) + 2]);
    t3 = ((V[(bx * 3) + 0] - V[(tx * 3) + 0]) * O[(tx * 3 + 1)] - (V[(bx * 3) + 1] - V[(tx * 3) + 1]) * O[(tx * 3) + 0]);
	radiika = powf(V[(bx * 3) + 0] - V[(tx * 3) + 0], 2) + pow(V[(bx * 3) + 1] - V[(tx * 3) + 1], 2) + pow(V[(bx * 3) + 2] - V[(tx * 3) + 2], 2);
	ssss = expf((-radiika)*((3.1416f * 2.0f) / (S[tx] * S[tx])));
	dssss_dr = expf(-((3.1416f * 2.0f) / (S[tx] * S[tx])));


	for (int i = 0; i<N; i++) {
		if (tx == i) { 					
			Vc[0] = Vc[0] + ssss* t1;
			Vc[1] = Vc[1] + ssss* t2;
			Vc[2] = Vc[2] + ssss* t3;

			dVx[0] = dssss_dr*(V[(bx * 3) + 0] - V[(tx * 3) + 0]) * t1 + dVx[0];
			dVx[1] = dssss_dr*(V[(bx * 3) + 1] - V[(tx * 3) + 1]) * t1 + O[tx * 3 + 2] * ssss + dVx[1];
			dVx[2] = dssss_dr*(V[(bx * 3) + 2] - V[(tx * 3) + 2]) * t1 - O[tx * 3 + 1] * ssss + dVx[2];

			dVy[0] = dssss_dr*(V[(bx * 3) + 0] - V[(tx * 3) + 0]) * t2 - O[tx * 3 + 2] * ssss + dVy[0];
			dVy[1] = dssss_dr*(V[(bx * 3) + 1] - V[(tx * 3) + 1]) * t2 + dVy[1];
			dVy[2] = dssss_dr*(V[(bx * 3) + 2] - V[(tx * 3) + 2]) * t2 + O[tx * 3 + 0] * ssss + dVy[2];

			dVz[0] = dssss_dr*(V[(bx * 3) + 0] - V[(tx * 3) + 0]) * t3 + O[tx * 3 + 1] * ssss + dVz[0];
			dVz[1] = dssss_dr*(V[(bx * 3) + 1] - V[(tx * 3) + 1]) * t3 - O[tx * 3 + 0] * ssss + dVz[1];
			dVz[2] = dssss_dr*(V[(bx * 3) + 2] - V[(tx * 3) + 2]) * t3 + dVz[2];
		}
		__syncthreads(); // barrier set to get all the threads to this point, before any further calculations
	}

	if (tx<3) {
		VN[(bx * 3) + tx] = V[(bx * 3) + tx] + 0.01f * Vc[tx];  // translates to global thread ids performing the tasks.
		domdt[tx] = dVx[0] * O[bx * 3 + 0] + dVx[1] * O[bx * 3 + 1] + dVx[2] * O[bx * 3 + 2];
		ON[bx * 3 + tx] = O[bx * 3 + tx] + domdt[tx] * 0.01f;
	}
}


__global__
 void AssignNewToOld(float* V, float* O, float* VN, float* ON, float* S, float* A, float* vx, float* E, float* S_m, float* Ss, int N) {
	//*A = 0.0f;  			// This kernel calculates the magnitude of angular velocity , Energy, Maximum speed and radius of vortex.
	//*vx = 0.0f;
	float Amagnit_old;
	float Amagnit_new;
	int tx = threadIdx.x;
	__shared__ float v[3];
	__shared__ float radiika;
	__shared__ float t1;
	__shared__ float Om22P;
	__shared__ float ssss;

	V[(tx * 3) + 0] = VN[(tx * 3) + 0];
	V[(tx * 3) + 1] = VN[(tx * 3) + 1];
	V[(tx * 3) + 2] = VN[(tx * 3) + 2];

	Amagnit_old = sqrtf(powf(O[tx * 3 + 0], 2) + powf(O[tx * 3 + 1], 2) + powf(O[tx * 3 + 2], 2));
	O[tx * 3 + 0] = ON[tx * 3 + 0];
	O[tx * 3 + 1] = ON[tx * 3 + 1];
	O[tx * 3 + 2] = ON[tx * 3 + 2];
	Amagnit_new = sqrtf(powf(O[tx * 3 + 0], 2) + powf(O[tx * 3 + 1], 2) + powf(O[tx * 3 + 2], 2));
	S[tx] = S[tx] * sqrtf(Amagnit_old / Amagnit_new);


	for (int i = 0; i<N; i++) {
		if (tx == i) {
			if (Amagnit_new >= (*A)) {
				*A = Amagnit_new;
				*E = (pow(Amagnit_new, 2)* pow(S[tx], 5));
				*S_m = Amagnit_new * S[tx];
				*Ss = S[tx];
			}
		}
		__syncthreads();
	}

	for(int i=0;i<N;i++){
		if(tx == i){
			v[0] = 0.5 - V[tx * 3 + 0];
			v[1] = 0.5 - V[tx * 3 + 1];
			v[2] = 0.5 - V[tx * 3 + 2];
			radiika = v[0] * v[0] + v[1] * v[1] + v[2] * v[2];
			t1 = v[1] * O[tx * 3 + 2] - v[2] * O[tx * 3 + 1];
			Om22P = (3.1416*2.0) / (S[tx] * S[tx]);
			ssss = expf(-radiika*Om22P);
			*vx = *vx + (ssss*t1);
		}
		__syncthreads();
	}
}

float randomgenerator()    		//Returns a random number between 0 and 1
{
	float x;
   	x=rand()/float(RAND_MAX);	    //Rand_MAX value depends on library, but is always less than 32767 on any standard library usage
   	return x;
}

float randomgenerator(float a, float b)  //Returns a random number between the passed parameters
{
	 float t;
     t=(b-a)*randomgenerator() + a;   //The values a and b (a>b) are the range in which the numbers are required.
	 return t;
}


int main(int argc, char ** argv) {

	clock_t start, end;
	start = clock();		// For time calculation
	int Ntime = 10000;        // Number of time steps
	float Delta_t = 0.01;  // Time step
	float Radius = 0.1;   // Radius of vorton (the same for all)
	int number = 700;                           // number of vortons
	int Ncout;
	float Replace;
	float StatisticalMoments[4];

	for (int i = 0; i<4; i++) {
		StatisticalMoments[i] = 0.0;
	}
	//--------------Define Host and device arrays------------------------//
	
	static float h_Vortex[2100];			// Arrays are defined for host and devices
	static float h_Omega_v[2100];
	static float h_VortexN[2100];
	static float h_Omega_vN[2100];
	static float h_Sigma[700];
	float h_Amagni;
	float h_Energy;
	float h_Speed_max;
	float h_Sigmas;
	float h_Vx;
	int size1 = sizeof(float);

	float *d_Vortex;
	float *d_Omega_v;
	float *d_VortexN;
	float *d_Omega_vN;
	float *d_Sigma;
	float *d_Amagni;
	float *d_Energy;
	float *d_Speed_max;
	float *d_Sigmas;
	float *d_Vx;

	hipMalloc((void **)&d_Vortex, number * 3 * size1);			// Size of variable is allocated on GPU's dram
	hipMalloc((void **)&d_Omega_v, number * 3 * size1);
	hipMalloc((void **)&d_VortexN, number * 3 * size1);
	hipMalloc((void **)&d_Omega_vN, number * 3 * size1);
	hipMalloc((void **)&d_Sigma, number*size1);
	hipMalloc((void **)&d_Amagni, size1);
	hipMalloc((void **)&d_Energy, size1);
	hipMalloc((void **)&d_Speed_max, size1);
	hipMalloc((void **)&d_Sigmas, size1);
	hipMalloc((void **)&d_Vx, size1);

	/*---------------------------------------Creating files------------------------------------------------*/

	//ofstream outfile1;
	//ofstream outfile2;
	// outfile1.open("./outputfiles/velocities_cuda.dat");
	// outfile2.open("./outputfiles/MaxValue_cuda.dat");

	/* ----------------------------------------Intilization of Arrays---------------------------------------*/

	for (int i = 0; i<number * 3; i++) { 			// Host arrays are initialized with a random function, giving them 	
		srand(i);									// values in the range of (0, 1)
		h_Vortex[i] = randomgenerator(1,0);
		srand(i);
		h_Omega_v[i] =randomgenerator(1,0)- 0.5;	
	}

	for (int i = 0; i<number; i++) {
		srand(i);
		h_Sigma[i] = Radius;
	}

	//----------------------------Transfer Data of host to Device -------------------//

	hipMemcpy(d_Vortex, h_Vortex, number * 3 * size1, hipMemcpyHostToDevice);	
	hipMemcpy(d_Omega_v, h_Omega_v, number * 3 * size1, hipMemcpyHostToDevice);	
	hipMemcpy(d_Sigma, h_Sigma, number*size1, hipMemcpyHostToDevice);				

	//--------------------------------------------------------------------------------------------------------//

	for (int itime = 0; itime<Ntime; itime++) {

		NewVortexDistrub << <number, number >> >(d_Vortex, d_Omega_v, d_VortexN, d_Omega_vN, d_Sigma, number); // kernel call with 
													// 700 blocks each with 700 threads.
		hipDeviceSynchronize();					// All threads required to hit a barrier before any further calculations.
		hipMemcpy(h_Omega_vN, d_Omega_vN, number * 3 * size1, hipMemcpyDeviceToHost); // Data transfer back to host array
		hipMemcpy(h_VortexN, d_VortexN, number * 3 * size1, hipMemcpyDeviceToHost);	// Data transfer back to host array

		/*--------------------------mapping to the cube back--------------------------------------*/
		Ncout = 0;
		for (int ivorton = 0; ivorton< number; ivorton++) {
			Replace = 0.0f;
			for (int i = 0; i <3; i++) {
				if (h_VortexN[ivorton * 3 + i]< 0.0f) { Replace = 1.0f; }
				if (h_VortexN[ivorton * 3 + i] > 1.0f) { Replace = 1.0f; }
			}
			if (Replace == 1.0) {
				Ncout = Ncout + 1;
				srand(ivorton);// intialization of co-ordinate of vortrons
				h_VortexN[ivorton * 3 + 0] = randomgenerator(1,0);
				h_VortexN[ivorton * 3 + 1] = randomgenerator(1,0);
				h_VortexN[ivorton * 3 + 2] = randomgenerator(1,0);
				// intilization of strength of vortrons
				h_Omega_vN[ivorton * 3 + 0] = randomgenerator(1,0) - 0.5;
				h_Omega_vN[ivorton * 3 + 1] = randomgenerator(1,0) - 0.5;
				h_Omega_vN[ivorton * 3 + 2] = randomgenerator(1,0) - 0.5;
				h_Sigma[ivorton] = Radius;   // intilization of radius of vortrons
			}
		}

		hipMemcpy(d_Omega_vN, h_Omega_vN, number * 3 * size1, hipMemcpyHostToDevice);
		hipMemcpy(d_VortexN, h_VortexN, number * 3 * size1, hipMemcpyHostToDevice);

		/*------------------------- mapping to the cube back---------------------------------------*/
		//the old parameters became new ones
		h_Amagni = 0.0f;
		h_Vx = 0.0f;
		hipMemcpy(d_Amagni,&h_Amagni,size1,hipMemcpyHostToDevice);
		hipMemcpy(d_Vx,&h_Vx,size1,hipMemcpyHostToDevice);
		AssignNewToOld <<<1, number >>>(d_Vortex, d_Omega_v, d_VortexN, d_Omega_vN, d_Sigma, d_Amagni, d_Vx, // 2nd kernel call for 
d_Energy, d_Speed_max, d_Sigmas, number);			// calculating magnitude of the arrays passed as parameters
		hipDeviceSynchronize();

		hipMemcpy(&h_Amagni, d_Amagni, size1, hipMemcpyDeviceToHost);			// Data Transfers made accordingly
		hipMemcpy(&h_Energy, d_Energy, size1, hipMemcpyDeviceToHost);
		hipMemcpy(&h_Speed_max, d_Speed_max, size1, hipMemcpyDeviceToHost);
		hipMemcpy(&h_Sigmas, d_Sigmas, size1, hipMemcpyDeviceToHost);
		hipMemcpy(&h_Vx, d_Vx, size1, hipMemcpyDeviceToHost);


		printf("%15.3f%15.3f%15.3e%15.3e%15.3e%15.3f", itime*Delta_t, h_Amagni, h_Energy, h_Speed_max, h_Sigmas, h_Vx);
		printf("\n");
		for (int ier = 0; ier<4; ier++) {
			StatisticalMoments[ier] = StatisticalMoments[ier] + pow(h_Vx, ier);
		}
	}

	end = clock();
	printf("Time taken for GPU code for 700X700 threads is %10.3f sec" , (float) (end-start)/CLOCKS_PER_SEC ); //compute time
	hipFree(d_Vortex);					// memory freed after the calculations
	hipFree(d_Omega_v);
	hipFree(d_VortexN);
	hipFree(d_Omega_vN);
	hipFree(d_Sigma);
	hipFree(d_Amagni);
	hipFree(d_Vx);
	hipFree(d_Energy);
	hipFree(d_Speed_max);
	hipFree(d_Sigmas);
	
	return 0;
}


